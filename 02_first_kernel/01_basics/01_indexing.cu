
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void whoami(void)
{
    int block_id =  blockIdx.x + 
                    blockIdx.y * gridDim.x + 
                    blockIdx.z * gridDim.x * gridDim.y; // gives us a unique identifier for each block in the grid

    int block_offset =  block_id * 
                        blockDim.x * blockDim.y * blockDim.z; // gives us the starting index of the threads in the block

    int thread_offset = threadIdx.x + 
                        threadIdx.y * blockDim.x + 
                        threadIdx.z * blockDim.x * blockDim.y; // gives us the starting index of the threads in the block

    int id = block_offset + thread_offset;

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
    // printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);

}

int main(int argc, char **argv)
{
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;

    int blocks_per_grid = (b_x * b_y * b_z);
    int threads_per_block = (t_x * t_y * t_z);
    
    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z);
    dim3 threadsPerBlock(t_x, t_y, t_z);

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}
